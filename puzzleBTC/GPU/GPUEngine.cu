#include "hip/hip_runtime.h"
#include "./../SECP256k1.h"

#include "GPUEngine.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <stdint.h>
#include "../hash/sha256.h" 
#include "../hash/ripemd160.h"
#include "../Timer.h"

#include "GPUMath.h"
#include "GPUHash.h"
#include "GPUBase58.h"

// #include "GPUCompute.h" 
//======================================================================================

#include <device_atomic_functions.h>
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__device__ uint64_t* _2Gnx = NULL;
__device__ uint64_t* _2Gny = NULL;

__device__ uint64_t* Gx = NULL;
__device__ uint64_t* Gy = NULL;

// ---------------------------------------------------------------------------------------

__device__ __noinline__ void Check__Hash(uint64_t* px, uint64_t* py, int32_t incr,
	uint32_t* hash160, uint32_t* out_found)
{	
	uint8_t isOdd = py[0] & 1; // 
	uint32_t _h[5];  

	_GetHash160Comp(px, isOdd, (uint8_t*)_h);  //---------- h = _h đây 

	uint32_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (_h[0] == hash160[0] &&
		_h[1] == hash160[1] &&
		_h[2] == hash160[2] &&
		_h[3] == hash160[3] &&
		_h[4] == hash160[4] 
	) {
  		uint32_t pos = atomicAdd(out_found, 1);
	
		if (pos < 1) {
			out_found[pos * ITEM_SIZE_A32 + 1] = tid;
			// out_found[pos * ITEM_SIZE_A32 + 2] = (uint32_t)(incr << 16) | (uint32_t)(mode << 15);// | (uint32_t)(endo);
			out_found[pos * ITEM_SIZE_A32 + 2] = (uint32_t)(incr << 16);// | (uint32_t)(endo);
			out_found[pos * ITEM_SIZE_A32 + 3] = _h[0];
			out_found[pos * ITEM_SIZE_A32 + 4] = _h[1];
			out_found[pos * ITEM_SIZE_A32 + 5] = _h[2];
			out_found[pos * ITEM_SIZE_A32 + 6] = _h[3]; 
			out_found[pos * ITEM_SIZE_A32 + 7] = _h[4];
		}
	}
}
#define CHECK__HASH(incr) Check__Hash(px, py, incr, hash160, out_found)


// GPUEngine.cu  
// //======================================================================================
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )

inline void __cudaSafeCall(hipError_t err, const char* file, const int line)
{
	if (hipSuccess != err){
		fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n", file, line, hipGetErrorString(err));
		exit(-1);
	}
	return;
} 
// ---------------------------------------------------------------------------------------

__global__ void compute_keys_comp_mode_sa(uint32_t* hash160, uint64_t* __inputKey, uint32_t* out_found)
{
			// blockDim.x = 128 --- <<<_,128>>> // blockIdx.x từ 0-> 48 
			// xPtr-yPtr : 0    - 512     // xPtr-yPtr : 1024 - 1536

	int xPtr = (blockIdx.x * blockDim.x) * 8;  
	int yPtr = xPtr + 4 * blockDim.x;

	uint64_t* startx = __inputKey + xPtr;  
	uint64_t* starty = __inputKey + yPtr;

	uint64_t dx[GRP_SIZE / 2 + 1][4];  //mảng để lưu giá trị delta x.
	uint64_t px[4]; 
	uint64_t py[4];
	uint64_t pyn[4];
	uint64_t sx[4];
	uint64_t sy[4];
	uint64_t dy[4];
	uint64_t _s[4];
	uint64_t _p[4]; //mảng để lưu các giá trị tạm thời.


	// Load starting key
	__syncthreads();    //Đồng bộ hóa các luồng trong block hiện tại. // __syncthreads() là một hàm đồng bộ hóa trong CUDA, để đồng bộ hóa tất cả các luồng trong một block. Khi gọi hàm này, tất cả các luồng trong block đó sẽ dừng lại cho đến khi tất cả các luồng đã đến điểm gọi hàm. Điều này đảm bảo rằng mọi phép toán trước đó trong block đã hoàn thành trước khi bất kỳ luồng nào tiếp tục thực hiện các phép toán tiếp theo.
	Load256A(sx, startx);
	Load256A(sy, starty);
	Load256(px, sx);
	Load256(py, sy);   // Tải các giá trị bắt đầu vào các mảng sx, sy, px, py

	// Fill group with delta x
	uint32_t i;
	for (i = 0; i < HSIZE; i++){ //HSIZE = (GRP_SIZE / 2 - 1) = 1023
		ModSub256(dx[i], Gx + 4 * i, sx);  
		}    // Tính toán các giá trị delta x cho nhóm điểm.
	ModSub256(dx[i], Gx + 4 * i, sx);   // For the first point
	ModSub256(dx[i + 1], _2Gnx, sx); // For the next center point

	_ModInvGrouped(dx);  // Compute modular inverse // Tính toán nghịch đảo modulo cho các giá trị delta x.

	// We use the fact that P + i*G and P - i*G has the same deltax, so the same inverse
	// We compute key in the positive and negative way from the center of the group

	// Check starting point
	CHECK__HASH(GRP_SIZE / 2); //GRP_SIZE = 1024*2  //  điểm khởi đầu.
	//-------CHECK__HASH(incr) Check__Hash(px, py, incr, hash160, out_found)
	
	ModNeg256(pyn, py);  // Tính giá trị âm của py

	
	//tính toán các giá trị x và y cho từng điểm 
	for (i = 0; i < HSIZE; i++) {   // HSIZE (GRP_SIZE / 2 - 1) = 1023 

		// P = StartPoint + i*G  //--- thay p2 = G // thay _p2 = _p
		Load256(px, sx);
		Load256(py, sy);
		ModSub256(dy, Gy + 4 * i, py);
				//--------------- hiiu... Secp256K1::NextKey  -------------------- 
		_ModMult(_s, dy, dx[i]);    //  s = (G.y-p1.y)*inverse(G.x-p1.x)
		_ModSqr(_p, _s);           // _p = pow2(s)

		ModSub256(px, _p, px);
		ModSub256(px, Gx + 4 * i);  // px = pow2(s) - p1.x - G.x; 

		ModSub256(py, Gx + 4 * i, px);
		_ModMult(py, _s);            // py = - s*(ret.x-G.x)
		ModSub256(py, Gy + 4 * i);   // py = - G.y - s*(ret.x-G.x);
				//-----------------------------------
 
		CHECK__HASH(GRP_SIZE / 2 + (i + 1));    
		//------CHECK__HASH(incr) Check__Hash(px, py, incr, hash160, out_found)

		// P = StartPoint - i*G, if (x,y) = i*G then (x,-y) = -i*G
		Load256(px, sx);   
		ModSub256(dy, pyn, Gy + 4 * i);
				//--------------- hiiu... Secp256K1::NextKey --------------------
		_ModMult(_s, dy, dx[i]);            //  s = (G.y-p1.y)*inverse(G.x-p1.x)
		_ModSqr(_p, _s);                   // _p = pow2(s)

		ModSub256(px, _p, px);
		ModSub256(px, Gx + 4 * i);          // px = pow2(s) - p1.x - G.x;

		ModSub256(py, px, Gx + 4 * i);
		_ModMult(py, _s);                   // py = s*(ret.x-G.x)
		ModSub256(py, Gy + 4 * i, py);      // py = - G.y - s*(ret.x-G.x);
				//-----------------------------------

		CHECK__HASH(GRP_SIZE / 2 - (i + 1));   
		//------CHECK__HASH(incr) Check__Hash(px, py, incr, hash160, out_found)
	}

	// First point (startP - (GRP_SZIE/2)*G)
	Load256(px, sx);
	Load256(py, sy);
	ModNeg256(dy, Gy + 4 * i);
	ModSub256(dy, py);

	_ModMult(_s, dy, dx[i]);              //  s = (G.y-p1.y)*inverse(G.x-p1.x)
	_ModSqr(_p, _s);                     // _p = pow2(s)

	ModSub256(px, _p, px);
	ModSub256(px, Gx + 4 * i);            // px = pow2(s) - p1.x - G.x;

	ModSub256(py, px, Gx + 4 * i);
	_ModMult(py, _s);                     // py = s*(ret.x-G.x)
	ModSub256(py, Gy + 4 * i, py);        // py = - G.y - s*(ret.x-G.x);

	
	CHECK__HASH(0);   //Kiểm tra hash cho điểm cuối cùng.
	//CHECK__HASH(incr) Check__Hash(px, py, incr, hash160, out_found)
	i++;

	// Next start point (startP +  *G) m //Cuối cùng, các giá trị x và y mới được lưu trở lại startx và starty
	Load256(px, sx);
	Load256(py, sy);
	ModSub256(dy, _2Gny, py);

	_ModMult(_s, dy, dx[i]);             //  s = (G.y-p1.y)*inverse(G.x-p1.x)
	_ModSqr(_p, _s);                    // _p = pow2(s)

	ModSub256(px, _p, px);
	ModSub256(px, _2Gnx);                // px = pow2(s) - p1.x - G.x;

	ModSub256(py, _2Gnx, px);
	_ModMult(py, _s);                    // py = - s*(ret.x-G.x)
	ModSub256(py, _2Gny);                // py = - G.y - s*(ret.x-G.x);

	// Update starting point
	__syncthreads();
	Store256A(startx, px);
	Store256A(starty, py);
}

// ---------------------------------------------------------------------------------------

int _ConvertSMVer2Cores(int major, int minor)
{
	// Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
	typedef struct {
		int SM;  // 0xMm (hexidecimal notation), M = SM Major version, 	// and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] = {
		{0x20, 32}, // Fermi Generation (SM 2.0) GF100 class
		{0x21, 48}, // Fermi Generation (SM 2.1) GF10x class
		{0x30, 192},
		{0x32, 192},
		{0x35, 192},
		{0x37, 192},
		{0x50, 128},
		{0x52, 128},
		{0x53, 128},
		{0x60,  64},
		{0x61, 128},
		{0x62, 128},
		{0x70,  64},
		{0x72,  64},
		{0x75,  64},
		{0x80,  64},
		{0x86, 128},
		{-1, -1}
	};

	int index = 0;

	while (nGpuArchCoresPerSM[index].SM != -1) {
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {return nGpuArchCoresPerSM[index].Cores;	}
		index++;
	}
	return 0;
}

// ----------------------------------------------------------------------------

GPUEngine::GPUEngine(Secp256K1* secp, int nbThreadGroup, int nbThreadPerGroup, int gpuId, const uint32_t* hashORxpoint)
{
	// Initialise CUDA
	this->nbThreadPerGroup = nbThreadPerGroup;

	initialised = false;

	int deviceCount = 0;
	CudaSafeCall(hipGetDeviceCount(&deviceCount));

	CudaSafeCall(hipSetDevice(gpuId));

	hipDeviceProp_t deviceProp;
	CudaSafeCall(hipGetDeviceProperties(&deviceProp, gpuId));

	if (nbThreadGroup == -1){ nbThreadGroup = deviceProp.multiProcessorCount * 8; } 

	this->nbThread = nbThreadGroup * nbThreadPerGroup;
	this->outputSize = (1 * ITEM_SIZE_A + 4);

	char tmp[512];
	sprintf(tmp, "GPU #%d %s (%dx%d cores) Grid(%dx%d) \n",
		gpuId, deviceProp.name, deviceProp.multiProcessorCount,
		_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
		nbThread / nbThreadPerGroup,
		nbThreadPerGroup);
	
	deviceName = std::string(tmp);

	// Prefer L1 (We do not use __shared__ at all)
	CudaSafeCall(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

	size_t stackSize = 49152; 
	CudaSafeCall(hipDeviceSetLimit(hipLimitStackSize, stackSize));

	// Allocate memory
	CudaSafeCall(hipMalloc((void**)&inputKey, nbThread * 32 * 2));
	CudaSafeCall(hipHostAlloc(&inputKeyPinned, nbThread * 32 * 2, hipHostMallocWriteCombined | hipHostMallocMapped));

	CudaSafeCall(hipMalloc((void**)&outputBuffer, outputSize));
	CudaSafeCall(hipHostAlloc(&outputBufferPinned, outputSize, hipHostMallocWriteCombined | hipHostMallocMapped));

	int K_SIZE = 5;

	CudaSafeCall(hipMalloc((void**)&inputHashORxpoint, K_SIZE * sizeof(uint32_t)));
	CudaSafeCall(hipHostAlloc(&inputHashORxpointPinned, K_SIZE * sizeof(uint32_t), hipHostMallocWriteCombined | hipHostMallocMapped));

	memcpy(inputHashORxpointPinned, hashORxpoint, K_SIZE * sizeof(uint32_t));

	CudaSafeCall(hipMemcpy(inputHashORxpoint, inputHashORxpointPinned, K_SIZE * sizeof(uint32_t), hipMemcpyHostToDevice));
	CudaSafeCall(hipHostFree(inputHashORxpointPinned));
	inputHashORxpointPinned = NULL;

	// generator table
	InitGenratorTable(secp);

	CudaSafeCall(hipGetLastError());

	initialised = true;

}

// ----------------------------------------------------------------------------

void GPUEngine::InitGenratorTable(Secp256K1* secp)
{
	// generator table
	uint64_t* _2GnxPinned;
	uint64_t* _2GnyPinned;

	uint64_t* GxPinned;
	uint64_t* GyPinned;

	uint64_t size = (uint64_t)GRP_SIZE;

	CudaSafeCall(hipMalloc((void**)&__2Gnx, 4 * sizeof(uint64_t)));
	CudaSafeCall(hipHostAlloc(&_2GnxPinned, 4 * sizeof(uint64_t), hipHostMallocWriteCombined | hipHostMallocMapped));

	CudaSafeCall(hipMalloc((void**)&__2Gny, 4 * sizeof(uint64_t)));
	CudaSafeCall(hipHostAlloc(&_2GnyPinned, 4 * sizeof(uint64_t), hipHostMallocWriteCombined | hipHostMallocMapped));

	size_t TSIZE = (size / 2) * 4 * sizeof(uint64_t);
	CudaSafeCall(hipMalloc((void**)&_Gx, TSIZE));
	CudaSafeCall(hipHostAlloc(&GxPinned, TSIZE, hipHostMallocWriteCombined | hipHostMallocMapped));

	CudaSafeCall(hipMalloc((void**)&_Gy, TSIZE));
	CudaSafeCall(hipHostAlloc(&GyPinned, TSIZE, hipHostMallocWriteCombined | hipHostMallocMapped));


	Point* Gn = new Point[size];
	Point g = secp->G;
	Gn[0] = g;
	g = secp->DoubleDirect(g); 
	Gn[1] = g;
	for (int i = 2; i < size; i++) {
		g = secp->AddDirect(g, secp->G);
		Gn[i] = g;
	}
	// _2Gn = CPU_GRP_SIZE*G   
	Point _2Gn = secp->DoubleDirect(Gn[size / 2 - 1]);

	int nbDigit = 4;
	for (int i = 0; i < nbDigit; i++) {
		_2GnxPinned[i] = _2Gn.x.bits64[i];
		_2GnyPinned[i] = _2Gn.y.bits64[i];
	}
	for (int i = 0; i < size / 2; i++) {
		for (int j = 0; j < nbDigit; j++) {
			GxPinned[i * nbDigit + j] = Gn[i].x.bits64[j];
			GyPinned[i * nbDigit + j] = Gn[i].y.bits64[j];
		}
	}

	delete[] Gn;

	CudaSafeCall(hipMemcpy(__2Gnx, _2GnxPinned, 4 * sizeof(uint64_t), hipMemcpyHostToDevice));
	CudaSafeCall(hipHostFree(_2GnxPinned));
	_2GnxPinned = NULL;

	CudaSafeCall(hipMemcpy(__2Gny, _2GnyPinned, 4 * sizeof(uint64_t), hipMemcpyHostToDevice));
	CudaSafeCall(hipHostFree(_2GnyPinned));
	_2GnyPinned = NULL;

	CudaSafeCall(hipMemcpy(_Gx, GxPinned, TSIZE, hipMemcpyHostToDevice));
	CudaSafeCall(hipHostFree(GxPinned));
	GxPinned = NULL;

	CudaSafeCall(hipMemcpy(_Gy, GyPinned, TSIZE, hipMemcpyHostToDevice));
	CudaSafeCall(hipHostFree(GyPinned));
	GyPinned = NULL;

	//hipMemcpyToSymbol : để sao chép dữ liệu từ bộ nhớ của máy chủ (host) vào bộ nhớ của thiết bị (device) cho các biến toàn cục (global variables)
	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(_2Gnx), &__2Gnx, sizeof(uint64_t*)));
	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(_2Gny), &__2Gny, sizeof(uint64_t*)));
	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(Gx), &_Gx, sizeof(uint64_t*)));
	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(Gy), &_Gy, sizeof(uint64_t*)));

}

// ----------------------------------------------------------------------------

int GPUEngine::GetGroupSize()
{	
	return GRP_SIZE; //GRP_SIZE = 1024*2
}

// ----------------------------------------------------------------------------

void GPUEngine::PrintCudaInfo()
{
	printf("GPUEngine::PrintCudaInfo() : ");
	const char* sComputeMode[] = {
		"Multiple host threads",
		"Only one host thread",
		"No host thread",
		"Multiple process threads",
		"Unknown",
		NULL
	};

	int deviceCount = 0;
	CudaSafeCall(hipGetDeviceCount(&deviceCount));

	for (int i = 0; i < deviceCount; i++) {
		CudaSafeCall(hipSetDevice(i));
		hipDeviceProp_t deviceProp;
		CudaSafeCall(hipGetDeviceProperties(&deviceProp, i));
		printf("GPU #%d %s (%dx%d cores) (Cap %d.%d) (%.1f MB) (%s)\n",
			i, deviceProp.name, deviceProp.multiProcessorCount,
			_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
			deviceProp.major, deviceProp.minor, (double)deviceProp.totalGlobalMem / 1048576.0,
			sComputeMode[deviceProp.computeMode]);
	}
}

// ----------------------------------------------------------------------------

GPUEngine::~GPUEngine()
{
	CudaSafeCall(hipFree(inputKey));
	CudaSafeCall(hipFree(inputHashORxpoint));
	CudaSafeCall(hipHostFree(outputBufferPinned));
	CudaSafeCall(hipFree(outputBuffer));
	CudaSafeCall(hipFree(__2Gnx));
	CudaSafeCall(hipFree(__2Gny));
	CudaSafeCall(hipFree(_Gx));
	CudaSafeCall(hipFree(_Gy));
}

// ----------------------------------------------------------------------------

int GPUEngine::GetNbThread() 
{
	return nbThread; 
}

// ----------------------------------------------------------------------------

bool GPUEngine::SetKeys(Point* p) //p ở đây có dạng (x=, y= , z=1)
{
	// Sets the starting keys for each thread 	// p must contains nbThread public keys
	for (int i = 0; i < nbThread; i += nbThreadPerGroup) { //nbThread = 6144 -- nbThreadPerGroup = 128 

		for (int j = 0; j < nbThreadPerGroup; j++) {

			inputKeyPinned[8 * i + j + 0 * nbThreadPerGroup] = p[i + j].x.bits64[0];
			inputKeyPinned[8 * i + j + 1 * nbThreadPerGroup] = p[i + j].x.bits64[1];
			inputKeyPinned[8 * i + j + 2 * nbThreadPerGroup] = p[i + j].x.bits64[2];
			inputKeyPinned[8 * i + j + 3 * nbThreadPerGroup] = p[i + j].x.bits64[3];

			inputKeyPinned[8 * i + j + 4 * nbThreadPerGroup] = p[i + j].y.bits64[0];
			inputKeyPinned[8 * i + j + 5 * nbThreadPerGroup] = p[i + j].y.bits64[1];
			inputKeyPinned[8 * i + j + 6 * nbThreadPerGroup] = p[i + j].y.bits64[2];
			inputKeyPinned[8 * i + j + 7 * nbThreadPerGroup] = p[i + j].y.bits64[3];
  		}
	}

	// Fill device memory
	CudaSafeCall(hipMemcpy(inputKey, inputKeyPinned, nbThread * 32 * 2, hipMemcpyHostToDevice));

	CudaSafeCall(hipHostFree(inputKeyPinned));
	inputKeyPinned = NULL;

	CudaSafeCall(hipMemset(outputBuffer, 0, 4));

	compute_keys_comp_mode_sa <<< nbThread / nbThreadPerGroup, nbThreadPerGroup >>>(inputHashORxpoint, inputKey, outputBuffer);
	return true;
}

// ----------------------------------------------------------------------------

bool GPUEngine::LaunchSEARCH_MODE_SA(std::vector<ITEM>& dataFound) 
{
	dataFound.clear();

	// Get the result
	hipEvent_t evt;
	CudaSafeCall(hipEventCreate(&evt));
	CudaSafeCall(hipMemcpyAsync(outputBufferPinned, outputBuffer, 4, hipMemcpyDeviceToHost, 0));
	CudaSafeCall(hipEventRecord(evt, 0));

	//The function enters a loop where it checks if the event has completed. //If not, it sleeps for 1 millisecond to avoid busy-waiting.
	while (hipEventQuery(evt) == hipErrorNotReady) {  	
		Timer::SleepMillis(1);// Sleep 1 ms to free the CPU 
	}
	
	CudaSafeCall(hipEventDestroy(evt));

	// Look for data found
	uint32_t nbFound = outputBufferPinned[0];

	// When can perform a standard copy, the kernel is eneded 
	CudaSafeCall(hipMemcpy(outputBufferPinned, outputBuffer, nbFound * ITEM_SIZE_A + 4, hipMemcpyDeviceToHost)); // ITEM_SIZE_A = 28

	for (uint32_t i = 0; i < nbFound; i++) //if found right key-hash-addr
	{ 
		uint32_t* itemPtr = outputBufferPinned + (i * ITEM_SIZE_A32 + 1); //ITEM_SIZE_A32 = 7
		ITEM it;
		it.thId = itemPtr[0];
		int16_t* ptr = (int16_t*)&(itemPtr[1]);
		//it.endo = ptr[0] & 0x7FFF;
		// it.mode = (ptr[0] & 0x8000) != 0;
		it.incr = ptr[1];  
		it.hash = (uint8_t*)(itemPtr + 2);
		dataFound.push_back(it);
	}
	   
	CudaSafeCall(hipMemset(outputBuffer, 0, 4));

	compute_keys_comp_mode_sa <<< nbThread / nbThreadPerGroup, nbThreadPerGroup >>>(inputHashORxpoint, inputKey, outputBuffer);

	return true;
}
